
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define DEFAULT_ROW  16384
#define DEFAULT_COL  16384
// time stamp function in seconds
double getTimeStamp() {
	struct timeval tv ;
	gettimeofday( &tv, NULL ) ;
	return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}
// host side matrix addition
void h_addmat(float *A, float *B, float *C, int nx, int ny){ 
	float* ia = A, *ib =B, *ic =C;
	for (int iy =0; iy<ny; iy++){
		for (int ix =0; ix<nx; ix++){
			
			ic[ix] = ia[ix] + ib[ix];
			//if (iy*nx + ix == 67133440) printf("the addition in host: %.6f + %.6f = %.6f\n",ia[ix],ib[ix],ic[ix]);
			
		}
		ia += nx;
		ib += nx;
		ic += nx;
	}
 }
//host side matrix comparison
int h_compareResult(float *h_C, float *d_C, int noElems){ 
	float *host_c = h_C,*device_c = d_C;
	for (int i =0; i<noElems; i++){
		if (*(host_c) != *(device_c)){
#ifdef DEBUG

			printf("the i = %d\n", i);
			printf("the data of CPU is %.6f\n", *(host_c));
			printf("the data of GPU is %.6f\n", *(device_c));

#endif
			return 1;
		} 
		host_c++;
		device_c++;
	}
	return 0;
 }
// device-side matrix addition
__global__ void f_addmat( float *A, float *B, float *C, int nx, int ny ){
	// kernel code might look something like this
	// but you may want to pad the matrices and index into them accordingly
	int ix = threadIdx.x + blockIdx.x*blockDim.x ;
	int iy = threadIdx.y + blockIdx.y*blockDim.y ;
	int idx = iy*nx + ix ;
	if( (ix<nx) && (iy<ny) )
	C[idx] = A[idx] + B[idx] ;
	//if (idx%2100000 == 0) printf("the addition in device: %.6f + %.6f = %.6f\n",A[idx],B[idx],C[idx]);
}


void initData(float* add, int noElems){
	int i;
	float a = 5.0;
	for (i=0; i< noElems; i++){
		*(add++) = ((float)rand()/(float)(RAND_MAX)) * a;
	}

}

int main(int argc, char* argv[]){

	if(argc != 3){
		printf("Error: wrong number of argument\n");
		exit(0);
	}

	int nx = atoi(argv[1]);
	int ny = atoi(argv[2]);

	int noElems = nx * ny;
	int bytes = noElems * sizeof(float);
#ifdef DEBUG
	printf("the input row # is %d\n",nx);
	printf("the input col # is %d\n",ny);
	printf("the noElems is %d\n",noElems);
	printf("the bytes is %d\n",bytes);
#endif
	// padding


	// alloc memeory host-side
	float *h_A;// = (float*) malloc(bytes);
	float *h_B;// = (float*) malloc(bytes);
	float *h_dC;// = (float*) malloc(bytes);	 //gpu result
	float *h_hC = (float*) malloc(bytes); // host result
	//float *h_dC;

	hipHostAlloc((void**)&h_A, bytes, 0);	
	hipHostAlloc((void**)&h_B, bytes, 0);		
	hipHostAlloc((void**)&h_dC, bytes, 0);	

	// init matrices with random data
	initData(h_A, noElems);
	initData(h_B, noElems);

	//alloc memeory device-side
	float *d_A, *d_B, *d_C;
	hipMalloc( &d_A, bytes);
	hipMalloc( &d_B, bytes);
	hipMalloc( &d_C, bytes);


	double timeStampA = getTimeStamp() ;


	// invoke Kernel
	int block_x, block_y = 1;
	if (nx < 1024){
		
		block_x = nx;
		while ((ny + block_y-1)/block_y > 65535){
			block_y ++;
		}
		while (block_x * block_y > 1024){
			block_x --;
		}
	}
	else{
		block_x = 1024;
	}

	int i;
	int guessBytesPerStream = 80000000;
	int bytesPerStream = guessBytesPerStream - guessBytesPerStream % (nx * sizeof(float));
	//bytesPerStream = bytesPerStream * nx;
	// each stream is at least 2.1 MBytes big to get performance 
/*
	while (bytesPerStream < guessBytesPerStream){
		bytesPerStream = bytesPerStream + nx*sizeof(float);
	}
*/
	int NSTREAMS = bytes/bytesPerStream;
	int remainBytes = bytes%bytesPerStream;
	hipStream_t stream[NSTREAMS+1];

	dim3 block( block_x, block_y ) ; // you will want to configure this
	dim3 grid( (nx + block.x-1)/block.x, (bytesPerStream/(sizeof(float) * nx) + block.y-1)/block.y ) ;
	printf("the number of stream is = %d\n", NSTREAMS);
#ifdef DEBUG
	printf("the final bytesPerStream is = %d\n", bytesPerStream);

	printf("the remainBytes is = %d\n", remainBytes);
	printf("the final block size is x = %d and y = %d \n",block_x, block_y);
	printf("the final grid dimension is x = %d and y = %d \n",(nx + block_x-1)/block_x, (bytesPerStream/(sizeof(float) * nx) + block.y-1)/block.y ) ;
#endif
	double timeStampB = getTimeStamp() ;
	for(i = 1; i <=NSTREAMS; i++ ){
		hipStreamCreate(&stream[i]);
		int offset = (i-1) * bytesPerStream/4;

		hipMemcpyAsync(&d_A[offset],&h_A[offset],bytesPerStream, hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(&d_B[offset],&h_B[offset],bytesPerStream, hipMemcpyHostToDevice, stream[i]);

		f_addmat<<<grid, block,0,stream[i]>>>( &d_A[offset], &d_B[offset], &d_C[offset], nx, bytesPerStream/(sizeof(float) * nx) ) ;
		hipMemcpyAsync(&h_dC[offset],&d_C[offset],bytesPerStream, hipMemcpyDeviceToHost,stream[i]);
	}

	if(remainBytes != 0){
		int remainEle = remainBytes/4;
		hipStream_t last;
		hipStreamCreate(&last);
		int offset = NSTREAMS * bytesPerStream/4;
		hipMemcpyAsync(&d_A[offset],&h_A[offset],remainBytes, hipMemcpyHostToDevice, last);
		hipMemcpyAsync(&d_B[offset],&h_B[offset],remainBytes, hipMemcpyHostToDevice, last);

		dim3 grid( (nx + block.x-1)/block.x, (remainEle/nx + block.y-1)/block.y ) ;
#ifdef DEBUG
	printf("the final remain block size is x = %d and y = %d \n",block_x, block_y);
	printf("the final remain grid dimension is x = %d and y = %d \n",(nx + block_x-1)/block_x, (remainEle/nx + block.y-1)/block.y ) ;
#endif
		f_addmat<<<grid, block,0,last>>>( &d_A[offset], &d_B[offset], &d_C[offset], nx, remainEle/nx ) ;
	
		hipMemcpyAsync(&h_dC[offset],&d_C[offset],remainBytes, hipMemcpyDeviceToHost,last);
		hipStreamSynchronize(last);
	}

	for(i = 1; i <=NSTREAMS; i++ ){
		hipStreamSynchronize(stream[i]);
	}

	hipDeviceSynchronize() ;

	double timeStampC = getTimeStamp() ;
	double timeStampD = getTimeStamp() ;
	// free GPU resources
	hipFree( d_A ) ; hipFree( d_B ) ; hipFree( d_C ) ;

	// check result
	h_addmat( h_A, h_B, h_hC, nx, ny ) ;
	// h_dC == h+hC???

#ifdef DEBUG
	float *ptr;
	ptr = h_dC;
	int n = 268419072;
	ptr = ptr + n;
	printf("the data of GPU at index %d before comparison is %.6f\n", n,*(ptr));
#endif
	if (h_compareResult(h_hC,h_dC,noElems) == 1){
			printf("the two results don't match\n");
	}
	else{
		printf("totoal= %.6f CPU_GPU_transfer = %.6f kernel =%.6f GPU_CPU_transfer= %.6f\n",timeStampD - timeStampA,timeStampB - timeStampA, timeStampC - timeStampB, timeStampD - timeStampC  );
		//printf("CPU_GPU_transfer_time = %.6f\n",timeStampB - timeStampA );
		//printf("kernel_time = %.6f\n",timeStampC - timeStampB );
		//printf("GPU_CPU_transfer_time = %.6f\n",timeStampD - timeStampC );
	}

	free(h_hC);
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_dC);
	hipDeviceReset() ;

}

