
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define DEFAULT_ROW  16384
#define DEFAULT_COL  16384
#define offset 1
// time stamp function in seconds
double getTimeStamp() {
	struct timeval tv ;
	gettimeofday( &tv, NULL ) ;
	return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}
// host side matrix addition
void h_addmat(float *A, float *B, float *C, int nx, int ny){ 
	float* ia = A, *ib =B, *ic =C;
	for (int iy =0; iy<ny - 1; iy++){
		if(iy == ny - 1) ic[0] = ia[0] + ib[0];
		else{
			for (int ix =0; ix<nx; ix++){
				
				ic[ix] = ia[ix] + ib[ix];
				//if (iy*nx + ix == 0) printf("the addition at index 0 in host: %.6f + %.6f = %.6f\n",ia[ix],ib[ix],ic[ix]);
				
			}
			ia += nx;
			ib += nx;
			ic += nx;
		}
	}
	
 }
//host side matrix comparison
int h_compareResult(float *h_C, float *d_C, int noElems){ 
	float *host_c = h_C,*device_c = d_C;
	for (int i =0; i<noElems; i++){
		if (*(host_c) != *(device_c)){
#ifdef DEBUG

			printf("the i = %d\n", i);
			printf("the data of CPU is %.6f\n", *(host_c));
			printf("the data of GPU is %.6f\n", *(device_c));

#endif
			return 1;
		} 
		host_c++;
		device_c++;
	}
	return 0;
 }
// device-side matrix addition
__global__ void f_addmat( float *A, float *B, float *C, int nx, int ny ){
	// kernel code might look something like this
	// but you may want to pad the matrices and index into them accordingly
	int ix = threadIdx.x + blockIdx.x*blockDim.x ;
	int iy = threadIdx.y + blockIdx.y*blockDim.y ;
	int idx = iy*nx + ix ;
	
	if( (ix<nx) && (iy<ny) )
	C[idx+offset] = A[idx+offset] + B[idx+offset] ;
#ifdef DEBUG
	if (idx == 1) printf("the addition when idenx = %d in device: %.6f + %.6f = %.6f\n",idx,A[idx],B[idx],C[idx]);
	if (idx%10001 == 0)printf("the addition when idenx = %d in device: %.6f + %.6f = %.6f\n",idx,A[idx],B[idx],C[idx]);
#endif
}

void initData(float* add, int noElems){
	int i;
	float a = 5.0;
	for (i=0; i< noElems; i++){
		*(add++) = ((float)rand()/(float)(RAND_MAX)) * a;
		//*(add++) = (float)i;
	}

}

int main(int argc, char* argv[]){

	if(argc != 3){
		printf("Error: wrong number of argument\n");
		exit(0);
	}

	int nx = atoi(argv[1]);
	int ny = atoi(argv[2]);

	int noElems = nx * ny;
	int bytes = noElems * sizeof(float);
#ifdef DEBUG
	printf("the input row # is %d\n",nx);
	printf("the input col # is %d\n",ny);
	printf("the noElems is %d\n",noElems);
	printf("the bytes is %d\n",bytes);
#endif
	// padding


	// alloc memeory host-side
	float *h_A = (float*) malloc(bytes);
	float *h_B = (float*) malloc(bytes);
	float *h_dC = (float*) malloc(bytes);	 //gpu result
	float *h_hC = (float*) malloc(bytes); // host result
	//float *h_dC;
	//cudaHostAlloc(&h_dC, bytes, 0);	


	// init matrices with random data
	initData(h_A, noElems);
	initData(h_B, noElems);

	//alloc memeory device-side
	float *d_A, *d_B, *d_C;
	hipMalloc( &d_A, bytes+offset*sizeof(float));
	hipMalloc( &d_B, bytes+offset*sizeof(float));
	hipMalloc( &d_C, bytes+offset*sizeof(float));

	double timeStampA = getTimeStamp() ;

	
	//printf("the first element of A in device is %.6f\n", (d_A));
/*
	float *d_A_offset, *d_B_offset;
	d_A_offset = d_A;
	d_B_offset = d_B;
	d_A_offset++;
	d_B_offset++;
*/	
	//transfer data to dev
	hipMemcpy( (d_A+offset), h_A, bytes, hipMemcpyHostToDevice) ;
	hipMemcpy( (d_B+offset), h_B, bytes, hipMemcpyHostToDevice) ;
	// note that the transfers would be twice as fast if h_A and h_B
	// matrices are pinned
	//printf("the first element of A in device is %.6f\n", *(d_A_offset));

	double timeStampB = getTimeStamp() ;
	ny ++; //adding ny for the extra element.
	// invoke Kernel
	int block_x, block_y = 1;
	if (nx < 1024){
		
		block_x = nx;
		while ((ny + block_y-1)/block_y > 65535){
			block_y ++;
		}
		while (block_x * block_y > 1024){
			block_x --;
		}
	}
	else{
		block_x = 1024;
	}
#ifdef DEBUG
	printf("the final block size is x = %d and y = %d \n",block_x, block_y);
	printf("the final grid dimension is x = %d and y = %d \n",(nx + block_x-1)/block_x, (ny + block_y-1)/block_y);
#endif
	dim3 block( 1024, 1) ; // you will want to configure this
	dim3 grid( (nx + block.x-1)/block.x, (ny + block.y-1)/block.y ) ;
	f_addmat<<<grid, block>>>( d_A, d_B, d_C, nx, ny ) ;

	hipDeviceSynchronize() ;

	double timeStampC = getTimeStamp() ;

	//copy data back
	hipMemcpy( h_dC, (d_C+offset), bytes, hipMemcpyDeviceToHost ) ;
	double timeStampD = getTimeStamp() ;
	// free GPU resources
	hipFree( d_A ) ; hipFree( d_B ) ; hipFree( d_C ) ;
	hipDeviceReset() ;
	// check result
	h_addmat( h_A, h_B, h_hC, nx, ny ) ;
	// h_dC == h+hC???
	free(h_A);
	free(h_B);
#ifdef DEBUG
	float *ptr;
	ptr = h_dC;
	int n = 0;
	ptr = ptr + n;
	printf("the data of GPU at index %d before comparison is %.6f\n", n,*(ptr));
#endif
	
	if (h_compareResult(h_hC,h_dC,noElems) == 1){
			printf("the two results don't match\n");
	}
	else{
		//printf("totoal= %.6f CPU_GPU_transfer = %.6f kernel =%.6f GPU_CPU_transfer= %.6f\n",timeStampD - timeStampA,timeStampB - timeStampA, timeStampC - timeStampB, timeStampD - timeStampC  );
		printf("%.6f %.6f %.6f %.6f\n",timeStampD - timeStampA,timeStampB - timeStampA, timeStampC - timeStampB, timeStampD - timeStampC  );
		//printf("CPU_GPU_transfer_time = %.6f\n",timeStampB - timeStampA );
		//printf("kernel_time = %.6f\n",timeStampC - timeStampB );
		//printf("GPU_CPU_transfer_time = %.6f\n",timeStampD - timeStampC );
	}

	hipHostFree(h_hC);

}

