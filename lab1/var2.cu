
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define DEFAULT_ROW  16384
#define DEFAULT_COL  16384
// time stamp function in seconds
double getTimeStamp() {
	struct timeval tv ;
	gettimeofday( &tv, NULL ) ;
	return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}
// host side matrix addition
void h_addmat(float *A, float *B, float *C, int nx, int ny){ 
	float* ia = A, *ib =B, *ic =C;
	for (int iy =0; iy<ny; iy++){
		for (int ix =0; ix<nx; ix++){
			ic[ix] = ia[ix] + ib[ix];
		}
		ia += nx;
		ib += nx;
		ic += nx;
	}
 }
//host side matrix comparison
int h_compareResult(float *h_C, float *d_C, int noElems){ 
	float* host_c = h_C,*device_c = d_C;
	for (int i =0; i<noElems; i++){
		if (*(host_c) != *(device_c)){
			return 1;
		} 
		host_c++;
		device_c++;
	}
	return 0;
 }
// device-side matrix addition
__global__ void f_addmat( float *A, float *B, float *C, int nx, int ny ){
	// kernel code might look something like this
	// but you may want to pad the matrices and index into them accordingly
	int ix = threadIdx.x + blockIdx.x*blockDim.x ;
	int iy = threadIdx.y + blockIdx.y*blockDim.y ;
	int idx = iy*ny + ix ;
	if( (ix<nx) && (iy<ny) )
	C[idx] = A[idx] + B[idx] ;
}

void initData(float* add, int noElems){
	int i;
	for (i=0; i< noElems; i++){
		*(add++) = (float)rand()/(float)(RAND_MAX);
	}

}

int main(int argc, char* argv[]){

	if(argc != 3){
		printf("Error: wrong number of argument\n");
		exit(0);
	}

	int nx = atoi(argv[1]);
	int ny = atoi(argv[2]);



	int noElems = nx * ny;
	int bytes = noElems * sizeof(float);
	// padding


	// alloc memeory host-side
	float *h_A = (float*) malloc(bytes);
	float *h_B = (float*) malloc(bytes);
	float *h_hC = (float*) malloc(bytes); // host result
	float *h_dC = (float*) malloc(bytes);	 //gpu result

	// init matrices with random data
	initData(h_A, noElems);
	initData(h_B, noElems);

	//alloc memeory device-side
	float *d_A, *d_B, *d_C;
	hipMalloc( &d_A, bytes);
	hipMalloc( &d_B, bytes);
	hipMalloc( &d_C, bytes);

	double timeStampA = getTimeStamp() ;

	//transfer data to dev
	hipMemcpy( d_A, h_A, bytes, hipMemcpyHostToDevice) ;
	hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice) ;
	// note that the transfers would be twice as fast if h_A and h_B
	// matrices are pinned

	double timeStampB = getTimeStamp() ;

	// invoke Kernel
	dim3 block( 1, 1024 ) ; // you will want to configure this
	dim3 grid( (nx + block.x-1)/block.x, (ny + block.y-1)/block.y ) ;
	f_addmat<<<grid, block>>>( d_A, d_B, d_C, nx, ny ) ;

	hipDeviceSynchronize() ;

	double timeStampC = getTimeStamp() ;

	//copy data back
	hipMemcpy( h_dC, d_C, bytes, hipMemcpyDeviceToHost ) ;
	double timeStampD = getTimeStamp() ;
	// free GPU resources
	hipFree( d_A ) ; hipFree( d_B ) ; hipFree( d_C ) ;
	hipDeviceReset() ;
	// check result
	h_addmat( h_A, h_B, h_hC, nx, ny ) ;
	// h_dC == h+hC???
	if (h_compareResult(h_hC,h_dC,noElems) == 1){
		printf("the two results don't matcj");
	}
	else{
		printf("totoal time = %.6f\n",timeStampD - timeStampA );
		printf("CPU_GPU_transfer_time = %.6f\n",timeStampB - timeStampA );
		printf("kernel_time = %.6f\n",timeStampC - timeStampB );
		printf("GPU_CPU_transfer_time = %.6f\n",timeStampD - timeStampC );
	}

}

