
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>


// time stamp function in seconds
double getTimeStamp() {
    struct timeval tv ;
    gettimeofday( &tv, NULL ) ;
    return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}
// host side matrix calculation
void h_compute_result(float *A, float *B, int n){
	float* ia = A, *ib =B;

	for (int iz =1; iz<n-1; iz++){
		for (int iy =1; iy<n-1; iy++){
            for (int ix =1; ix<n-1; ix++){
			ia[iz*(n)*(n) + iy * (n) + ix] = 0.8 * (ib[iz*(n)*(n) + iy * n + ix + 1] + ib[iz*(n)*(n) + iy * n + ix - 1] +
                                                    ib[iz*(n)*(n) + (iy + 1) * n + ix] + ib[iz*(n)*(n) + (iy - 1) * n + ix] +
                                                    ib[(iz + 1)*(n)*(n) + iy * n + ix] + ib[(iz - 1)*(n)*(n) + iy * n + ix]);


		}
	}
 }
}

//host side matrix comparison

int h_compareResult(float *h_A, float *d_A, int noElems){
	float *host_a = h_A,*device_a = d_A;
	for (int i =0; i<noElems; i++){
		if (*(host_a) != *(device_a)){
#ifdef DEBUG

			printf("the i = %d\n", i);
			printf("the data of CPU is %.6f\n", *(host_a));
			printf("the data of GPU is %.6f\n", *(device_a));

#endif
			return 1;
		}
        host_a++;
        device_a++;
	}
	return 0;
 }

 __constant__ float B[500*500*500*sizeof(float)];
// device-side matrix addition
__global__ void f_jocobiRelaxation( float *A, int n ){
	// kernel code might look something like this
	// but you may want to pad the matrices and index into them accordingly
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;
	int idx = iz*n*n + iy*n + ix ;
	if( (ix<n-1) && (iy<n-1) && (iz<n-1) &&(ix>=1) && (iy>=1) && (iz>=1)){

        A[idx] =0.8 * ( B[idx+1] + B[idx - 1] + B[idx + n] + B[idx - n] + B[idx + n * n] + B[idx - n * n] );
#ifdef DEBUG
        printf("the addition at idx = %d in device: %.6f \n",idx, A[idx]);
        //if (idx == 111) printf("the addition at idx = %d in device: %.6f \n",idx, A[idx]);
#endif
	}
}

void initDataB(float* ib, int n){
    for (int iz =0; iz<n; iz++){
        for (int iy =0; iy<n; iy++){
            for (int ix =0; ix<n; ix++){
                ib[iz*(n)*(n) + iy * (n) + ix] = 1.1 * ( ix + iy + iz);
            }
        }
    }
}
void initDataA(float* ia, int noElem){
    for (int i=0; i<noElem; i++) {
        ia[i] = 0;
    }
}
float SumDataA(float* ia, int noElem){
    float r = 0;
    for (int i=0; i<noElem; i++) {
        r += ia[i]/10000;
    }
    return r;
}

int main(int argc, char* argv[]){

	if(argc != 2){
		printf("Error: wrong number of argument\n");
		exit(0);
	}

	int n = atoi(argv[1]);


	// do the input argument check.
	if(n<=2){
		printf("Error: input arguement can't be zero or negative\n");
		exit(0);
	}

	int noElems = n * n * n;
	int bytes = noElems * sizeof(float);
    printf("number of element is %d \n", noElems);


	// alloc memeory host-side
	float *h_hA  = (float*) malloc(bytes); // host result


	float *h_B; // host result
    float *h_dA;

	//pin memory in host side
	hipHostAlloc((void**)&h_B, bytes, 0);
    hipHostAlloc((void**)&h_dA, bytes, 0);

	// init matrices with random data
    initDataA(h_hA, noElems);
    initDataA(h_dA, noElems);
    initDataB(h_B, n);
	//alloc memory device-side
	float *d_A, *B;
	hipMalloc( &d_A, bytes);
	hipMalloc( &B, bytes);

    double timeStampA = getTimeStamp() ;

    // getting host side result
    h_compute_result( h_hA, h_B, n) ;

    double timeStampB = getTimeStamp() ;
    printf("time taken for cpu calculation = %lf ms. \n",(timeStampB - timeStampA) );




    double timeStampC = getTimeStamp() ;
    hipMemcpyToSymbol( HIP_SYMBOL(B), h_B, bytes) ;
    double timeStampD = getTimeStamp() ;
    printf("time taken for memory transfer = %lf micro s. \n",(timeStampD - timeStampC)*1000  );
    // invoke Kernel
    dim3 block( 32, 32, 1) ; // you will want to configure this
    dim3 grid( (n + block.x-1)/block.x, (n + block.y-1)/block.y, (n+ block.z-1)/block.z ) ;
    f_jocobiRelaxation<<<grid, block>>>( d_A, n);
    hipDeviceSynchronize() ;
    double timeStampE = getTimeStamp() ;
    printf("time taken for kernel = %lf ms. \n",(timeStampE - timeStampD)  );

    //copy data back
    hipMemcpy( h_dA, d_A, bytes, hipMemcpyDeviceToHost ) ;
    double timeStampF = getTimeStamp() ;
    printf("time taken for copy data back to host = %lf ms. \n",(timeStampF - timeStampE)  );
    if (h_compareResult(h_hA, h_dA, noElems) == 1){
        printf("Error: the two results don't match\n");
    }
    else{
        printf("the result match\n");
    }
    hipFree( d_A ) ; hipFree( B );
    hipDeviceReset() ;
    free(h_hA);
    hipHostFree(h_B);
    hipHostFree(h_dA);
/*
	int i;
	// calculating minimum bytes each Stream should take according to the calculated block_y
	
	int minimumBytesPerStream = nx * sizeof(float) * 4 * block_y;	
	while (minimumBytesPerStream < 4194304*16){	// 4194304 is when 1024(thread) * 2 (blocks/SMS) * 16 (SMS) * 4 (sizeof(float)) * 2 (Two float number required for addition), we want data transfer is multiple of this number
		minimumBytesPerStream = minimumBytesPerStream * 2;
	}
	// yPerStream is mutiple of 4 so every thread can process 4 different y in one stream
	//int yPerStream = minimumBytesPerStream/ nx;
	// calculating bytes each Stream according to the calculated yPerStream
	//int bytesPerStream = nx * sizeof(float) * yPerStream;
	// calculating number of Streams according to the calculated bytesPerStream
	//int NSTREAMS = bytes/bytesPerStream;
	// if there is data remain where they are not multiple of bytesPerStream
	//int remainBytes = bytes%bytesPerStream;
	// initialize the stream array
	//cudaStream_t stream[NSTREAMS+1];
	// input the pre-calculated block size and calculate the grid size
	//dim3 block( block_x, block_y ) ; // you will want to configure this
	//dim3 grid( (nx + block.x-1)/block.x, (bytesPerStream/(sizeof(float) * nx) + block.y-1)/block.y ) ;

#ifdef DEBUG
	printf("the final bytesPerStream is = %d\n", bytesPerStream);

	printf("the remainBytes is = %d\n", remainBytes);
	printf("the final block size is x = %d and y = %d \n",block_x, block_y);
	printf("the final grid dimension is x = %d and y = %d \n",(nx + block_x-1)/block_x, (yPerStream + block.y-1)/block.y );
#endif
	// initialize the event for calculating accumulate kernel time.
	// NOTE: if we don't need to calculating the accumulate kernel time, the total time is at least 10% faster.
	// But  kernel time is important to show. 
	cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);
	float timeStampA = getTimeStamp() ;
	float timeStampB= getTimeStamp() ;
	float milliseconds;
	float AccumulateKernelTime = 0;
	for(i = 1; i <=NSTREAMS; i++ ){
		// create stream
		cudaStreamCreate(&stream[i]);
		//calculating offset
		int offset = (i-1) * bytesPerStream/4;
		//Asynch copy data from host to device 
		cudaMemcpyAsync(&d_A[offset],&h_A[offset],bytesPerStream, cudaMemcpyHostToDevice, stream[i]);
		cudaMemcpyAsync(&d_B[offset],&h_B[offset],bytesPerStream, cudaMemcpyHostToDevice, stream[i]);
		//record the timestamp before kernel invoke
		cudaEventRecord(start);
		//invoke kernel
		f_addmat4<<<grid, block,0,stream[i]>>>( &d_A[offset], &d_B[offset], &d_C[offset], nx, bytesPerStream/(4* sizeof(float) * nx), bytesPerStream/(4* sizeof(float)) ) ;
		//record the timestamp before kernel invoke		
		cudaEventRecord(stop);	
		cudaEventSynchronize(stop);
		// write down the difference
		cudaEventElapsedTime(&milliseconds, start, stop);
		// add this time to accumulated time
		AccumulateKernelTime += milliseconds/1000;
		//Asynch copy data from device back to host 
		cudaMemcpyAsync(&h_dC[offset],&d_C[offset],bytesPerStream, cudaMemcpyDeviceToHost,stream[i]);
	}
	// if there is remaining byte, we do the process one more time
	if(remainBytes != 0){
		int remainEle = remainBytes/4;
		cudaStream_t last;
		cudaStreamCreate(&last);
		int offset = NSTREAMS * bytesPerStream/4;
		cudaMemcpyAsync(&d_A[offset],&h_A[offset],remainBytes, cudaMemcpyHostToDevice, last);
		cudaMemcpyAsync(&d_B[offset],&h_B[offset],remainBytes, cudaMemcpyHostToDevice, last);
		dim3 grid( (nx + block.x-1)/block.x, (remainEle/nx + block.y-1)/block.y ) ;
		cudaEventRecord(start);
		f_addmat<<<grid, block,0,last>>>( &d_A[offset], &d_B[offset], &d_C[offset], nx, remainEle/nx ) ;
		cudaEventRecord(stop);	
		cudaEventElapsedTime(&milliseconds, start, stop);
		AccumulateKernelTime += milliseconds/1000;
		cudaMemcpyAsync(&h_dC[offset],&d_C[offset],remainBytes, cudaMemcpyDeviceToHost,last);
		cudaStreamSynchronize(last);
	}

	float timeStampC = getTimeStamp() ;
	//wait for all stream finish the job
	for(i = 1; i <=NSTREAMS; i++ ){
		cudaStreamSynchronize(stream[i]);
	}
	
	cudaDeviceSynchronize() ;
	//time where device side jobs have been finished
	float timeStampD = getTimeStamp() ;

	// free some Host and GPU resources that are not needed anymore
	cudaFreeHost(h_A);
	cudaFreeHost(h_B);
	cudaFree( d_A ) ; cudaFree( d_B ) ; cudaFree( d_C ) ;

#ifdef DEBUG
	float * ptr;
	int n;
	ptr = h_dC;
	n = 0;
	ptr = ptr + n;
	printf("the data of GPU at index %d before comparison is %.6f\n", n,*(ptr));
#endif	
	//h_compareResult compares the result computed by host and result computed by device
	//if any element is not same, the function will return 1, otherwise print out the time 
	if (h_compareResult(h_hC,h_dC,noElems) == 1){
			printf("Error: the two results don't match\n");
	}
	else{
		//printf(" %.6f  %.6f %.6f %.6f\n",timeStampD - timeStampA,timeStampB - timeStampA, AccumulateKernelTime, timeStampD - timeStampC  );
		printf(" %.6f  %.6f %.6f %.6f\n",timeStampD - timeStampA,timeStampB - timeStampA, AccumulateKernelTime, timeStampD - timeStampC  );
	}
	// free rest Host Side Resources
	cudaFreeHost(h_dC);
	free(h_hC);
	cudaDeviceReset();
 */
}
