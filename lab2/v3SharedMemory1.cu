
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define blockX  256
#define blockY  1
#define blockZ 4

// time stamp function in seconds
double getTimeStamp() {
    struct timeval tv ;
    gettimeofday( &tv, NULL ) ;
    return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}
// host side matrix calculation
void h_compute_result(float *A, float *B, int n){
	float* ia = A, *ib =B;

	for (int iz =1; iz<n-1; iz++){
		for (int iy =1; iy<n-1; iy++){
            for (int ix =1; ix<n-1; ix++){
			ia[iz*(n)*(n) + iy * (n) + ix] = (ib[iz*(n)*(n) + iy * n + ix + 1] + ib[iz*(n)*(n) + iy * n + ix - 1] +
                                                    ib[iz*(n)*(n) + (iy + 1) * n + ix] + ib[iz*(n)*(n) + (iy - 1) * n + ix] +
                                                    ib[(iz + 1)*(n)*(n) + iy * n + ix] + ib[(iz - 1)*(n)*(n) + iy * n + ix]);


		}
	}
 }
}

//host side matrix comparison

int h_compareResult(float *h_A, float *d_A, int noElems){
	float *host_a = h_A,*device_a = d_A;
	for (int i =0; i<noElems; i++){
		if (*(host_a) != *(device_a)){
#ifdef DEBUG

			printf("the i = %d\n", i);
			printf("the data of CPU is %.6f\n", *(host_a));
			printf("the data of GPU is %.6f\n", *(device_a));

#endif
			return 1;
		}
        host_a++;
        device_a++;
	}
	return 0;
 }

// device-side matrix addition
__global__ void f_jocobiRelaxation( float *A, float *B, int n){
	// kernel code might look something like this
	// but you may want to pad the matrices and index into them accordingly
	__shared__ float block[1024];
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;
    int index_block = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int idx = iz*n*n + iy*n + ix ;
    //printf("the idx  = %d in device: %d \n",idx);
    block[index_block] = B[idx];
#ifdef DEBUG
    if (idx == 141){
        printf("at idx = %d in device, ix = %d, iy = %d, iz = %d \n",idx, idx, ix, iy, iz);
        printf("at idx = %d in device, threadx = %d, thready = %d, threadz = %d \n",idx, threadIdx.x, threadIdx.y, threadIdx.z);}
#endif
	__syncthreads();
    // resolving the multuple
    if (ix <= n && iy <= n && iz <= n){
        if( (ix<n-1) && (iy<n-1) && (iz<n-1) &&(ix>=1) && (iy>=1) && (iz>=1)) {
            if (threadIdx.x <= 1 || threadIdx.x >= blockDim.x - 2 || threadIdx.y <= 1 || threadIdx.y >= blockDim.y - 2 ||
                threadIdx.z <= 1 || threadIdx.z >= blockDim.z - 2) {
                A[idx] = (B[idx + 1] + B[idx - 1] + B[idx + n] + B[idx - n] + B[idx + n * n] + B[idx - n * n]);
                if (idx == 141) printf("edge \n");
            }
            else {
                A[idx] = (block[index_block + 1] + block[index_block - 1] + block[index_block + blockDim.x] + block[index_block - blockDim.x] +
                                block[index_block + blockDim.x * blockDim.y] + block[index_block - blockDim.x * blockDim.y]);
                if (idx == 141) printf("center \n");
            }
        }

#ifdef DEBUG
        //printf("the addition at idx = %d in device: %.6f \n",idx, A[idx]);
        if (idx == 141) printf("the addition at idx = %d in device: %.6f \n",idx, A[idx]);
//        if (idx == 118){
//            printf("at idx = %d in device, ix = %d, iy = %d, iz = %d \n",idx, threadIdx.x, threadIdx.y, threadIdx.z);
//            printf("at idx = %d in device, threadx = %d, thready = %d, threadz = %d \n",idx, ix, iy, iz);
//
//            printf("the x+1 at idx = %d in device: %.6f \n",idx, block[index_block + 1]);
//            printf("the x-1 at idx = %d in device: %.6f \n",idx, block[index_block - 1]);
//            printf("the y+1 at idx = %d in device: %.6f \n",idx, block[index_block + blockDim.x * blockDim.y]);
//            printf("the y-1 at idx = %d in device: %.6f \n",idx, block[index_block - blockDim.x * blockDim.y]);
//            printf("the z+1 at idx = %d in device: %.6f \n",idx, block[index_block + blockDim.x * blockDim.y]);
//            printf("the z-1 at idx = %d in device: %.6f \n",idx, block[index_block - blockDim.x * blockDim.y]);
//        }

#endif
    }



}

void initDataB(float* ib, int n){
    for (int iz =0; iz<n; iz++){
        for (int iy =0; iy<n; iy++){
            for (int ix =0; ix<n; ix++){
                //ib[iz*(n)*(n) + iy * (n) + ix] = 1.1 * ( ix + iy + iz);
                ib[iz*(n)*(n) + iy * (n) + ix] = ( ix + iy*n + iz*n*n);
            }
        }
    }
}
void initDataA(float* ia, int noElem){
    for (int i=0; i<noElem; i++) {
        ia[i] = 0;
    }
}
float SumDataA(float* ia, int noElem){
    float r = 0;
    for (int i=0; i<noElem; i++) {
        r += ia[i]/10000;
    }
    return r;
}

int main(int argc, char* argv[]){

	if(argc != 2){
		printf("Error: wrong number of argument\n");
		exit(0);
	}

	int n = atoi(argv[1]);


	// do the input argument check.
	if(n<=2){
		printf("Error: input arguement can't be zero or negative\n");
		exit(0);
	}

	int noElems = n * n * n;
	int bytes = noElems * sizeof(float);
    //printf("number of element is %d \n", noElems);
    printf("x =  %d \n", blockX);
    printf("y =  %d \n", blockY);
    printf("z =  %d \n", blockZ);
	// alloc memeory host-side
	float *h_hA  = (float*) malloc(bytes); // host result


	float *h_B; // host result
    float *h_dA;

	//pin memory in host side
	hipHostAlloc((void**)&h_B, bytes, 0);
    hipHostAlloc((void**)&h_dA, bytes, 0);

	// init matrices with random data
    initDataA(h_hA, noElems);
    initDataA(h_dA, noElems);
    initDataB(h_B, n);
	//alloc memory device-side
	float *d_A, *d_B;
	hipMalloc( &d_A, bytes);
	hipMalloc( &d_B, bytes);

    double timeStampA = getTimeStamp() ;

    // getting host side result
    h_compute_result( h_hA, h_B, n) ;

    double timeStampB = getTimeStamp() ;
    printf("time taken for cpu calculation = %lf s. \n",(timeStampB - timeStampA) );




    double timeStampC = getTimeStamp() ;
    hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice) ;
    double timeStampD = getTimeStamp() ;
    printf("time taken for memory transfer = %lf ms \n",(timeStampD - timeStampC)*1000  );
    // invoke Kernel
    dim3 block( blockX, blockY, blockZ) ; // you will want to configure this
    dim3 grid( (n + block.x-1)/block.x, (n + block.y-1)/block.y, (n+ block.z-1)/block.z ) ;
    f_jocobiRelaxation<<<grid, block>>>( d_A, d_B, n);
    hipDeviceSynchronize() ;
    double timeStampE = getTimeStamp() ;
    printf("time taken for kernel = %lf ms. \n",(timeStampE - timeStampD)*1000   );

    //copy data back
    hipMemcpy( h_dA, d_A, bytes, hipMemcpyDeviceToHost ) ;
    double timeStampF = getTimeStamp() ;
    printf("time taken for copy data back to host = %lf ms. \n",(timeStampF - timeStampE)*1000   );
    if (h_compareResult(h_hA, h_dA, noElems) == 1){
        printf("Error: the two results don't match\n");
    }
    else{
        printf("the result match\n");
    }
    hipFree( d_A ) ; hipFree( d_B );
    hipHostFree(h_B);
    hipHostFree(h_dA);
    hipDeviceReset() ;
    free(h_hA);

/*
	int i;
	// calculating minimum bytes each Stream should take according to the calculated block_y
	
	int minimumBytesPerStream = nx * sizeof(float) * 4 * block_y;	
	while (minimumBytesPerStream < 4194304*16){	// 4194304 is when 1024(thread) * 2 (blocks/SMS) * 16 (SMS) * 4 (sizeof(float)) * 2 (Two float number required for addition), we want data transfer is multiple of this number
		minimumBytesPerStream = minimumBytesPerStream * 2;
	}
	// yPerStream is mutiple of 4 so every thread can process 4 different y in one stream
	//int yPerStream = minimumBytesPerStream/ nx;
	// calculating bytes each Stream according to the calculated yPerStream
	//int bytesPerStream = nx * sizeof(float) * yPerStream;
	// calculating number of Streams according to the calculated bytesPerStream
	//int NSTREAMS = bytes/bytesPerStream;
	// if there is data remain where they are not multiple of bytesPerStream
	//int remainBytes = bytes%bytesPerStream;
	// initialize the stream array
	//cudaStream_t stream[NSTREAMS+1];
	// input the pre-calculated block size and calculate the grid size
	//dim3 block( block_x, block_y ) ; // you will want to configure this
	//dim3 grid( (nx + block.x-1)/block.x, (bytesPerStream/(sizeof(float) * nx) + block.y-1)/block.y ) ;

#ifdef DEBUG
	printf("the final bytesPerStream is = %d\n", bytesPerStream);

	printf("the remainBytes is = %d\n", remainBytes);
	printf("the final block size is x = %d and y = %d \n",block_x, block_y);
	printf("the final grid dimension is x = %d and y = %d \n",(nx + block_x-1)/block_x, (yPerStream + block.y-1)/block.y );
#endif
	// initialize the event for calculating accumulate kernel time.
	// NOTE: if we don't need to calculating the accumulate kernel time, the total time is at least 10% faster.
	// But  kernel time is important to show. 
	cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);
	float timeStampA = getTimeStamp() ;
	float timeStampB= getTimeStamp() ;
	float milliseconds;
	float AccumulateKernelTime = 0;
	for(i = 1; i <=NSTREAMS; i++ ){
		// create stream
		cudaStreamCreate(&stream[i]);
		//calculating offset
		int offset = (i-1) * bytesPerStream/4;
		//Asynch copy data from host to device 
		cudaMemcpyAsync(&d_A[offset],&h_A[offset],bytesPerStream, cudaMemcpyHostToDevice, stream[i]);
		cudaMemcpyAsync(&d_B[offset],&h_B[offset],bytesPerStream, cudaMemcpyHostToDevice, stream[i]);
		//record the timestamp before kernel invoke
		cudaEventRecord(start);
		//invoke kernel
		f_addmat4<<<grid, block,0,stream[i]>>>( &d_A[offset], &d_B[offset], &d_C[offset], nx, bytesPerStream/(4* sizeof(float) * nx), bytesPerStream/(4* sizeof(float)) ) ;
		//record the timestamp before kernel invoke		
		cudaEventRecord(stop);	
		cudaEventSynchronize(stop);
		// write down the difference
		cudaEventElapsedTime(&milliseconds, start, stop);
		// add this time to accumulated time
		AccumulateKernelTime += milliseconds/1000;
		//Asynch copy data from device back to host 
		cudaMemcpyAsync(&h_dC[offset],&d_C[offset],bytesPerStream, cudaMemcpyDeviceToHost,stream[i]);
	}
	// if there is remaining byte, we do the process one more time
	if(remainBytes != 0){
		int remainEle = remainBytes/4;
		cudaStream_t last;
		cudaStreamCreate(&last);
		int offset = NSTREAMS * bytesPerStream/4;
		cudaMemcpyAsync(&d_A[offset],&h_A[offset],remainBytes, cudaMemcpyHostToDevice, last);
		cudaMemcpyAsync(&d_B[offset],&h_B[offset],remainBytes, cudaMemcpyHostToDevice, last);
		dim3 grid( (nx + block.x-1)/block.x, (remainEle/nx + block.y-1)/block.y ) ;
		cudaEventRecord(start);
		f_addmat<<<grid, block,0,last>>>( &d_A[offset], &d_B[offset], &d_C[offset], nx, remainEle/nx ) ;
		cudaEventRecord(stop);	
		cudaEventElapsedTime(&milliseconds, start, stop);
		AccumulateKernelTime += milliseconds/1000;
		cudaMemcpyAsync(&h_dC[offset],&d_C[offset],remainBytes, cudaMemcpyDeviceToHost,last);
		cudaStreamSynchronize(last);
	}

	float timeStampC = getTimeStamp() ;
	//wait for all stream finish the job
	for(i = 1; i <=NSTREAMS; i++ ){
		cudaStreamSynchronize(stream[i]);
	}
	
	cudaDeviceSynchronize() ;
	//time where device side jobs have been finished
	float timeStampD = getTimeStamp() ;

	// free some Host and GPU resources that are not needed anymore
	cudaFreeHost(h_A);
	cudaFreeHost(h_B);
	cudaFree( d_A ) ; cudaFree( d_B ) ; cudaFree( d_C ) ;

#ifdef DEBUG
	float * ptr;
	int n;
	ptr = h_dC;
	n = 0;
	ptr = ptr + n;
	printf("the data of GPU at index %d before comparison is %.6f\n", n,*(ptr));
#endif	
	//h_compareResult compares the result computed by host and result computed by device
	//if any element is not same, the function will return 1, otherwise print out the time 
	if (h_compareResult(h_hC,h_dC,noElems) == 1){
			printf("Error: the two results don't match\n");
	}
	else{
		//printf(" %.6f  %.6f %.6f %.6f\n",timeStampD - timeStampA,timeStampB - timeStampA, AccumulateKernelTime, timeStampD - timeStampC  );
		printf(" %.6f  %.6f %.6f %.6f\n",timeStampD - timeStampA,timeStampB - timeStampA, AccumulateKernelTime, timeStampD - timeStampC  );
	}
	// free rest Host Side Resources
	cudaFreeHost(h_dC);
	free(h_hC);
	cudaDeviceReset();
 */
}
